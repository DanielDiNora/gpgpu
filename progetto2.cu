#import <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <math.h>

void error(char const *str)
{
	fprintf(stderr, "%s\n", str);
	exit(1);
}

void cuda_check(hipError_t err, char const *str)
{
	if (err != hipSuccess) {
		fprintf(stderr, "%s: CUDA error %d (%s)\n",
			str, err, hipGetErrorString(err));
	}
}

__host__ __device__
float4 operator+(const float4 &a, const float4 &b)
{
	return make_float4(a.x + b.x, a.y + b.y, a.z + b.z, a.w + b.w);
}

__host__ __device__
float4 operator-(const float4 &a, const float4 &b)
{
	return make_float4(a.x - b.x, a.y -b.y, a.z - b.z, a.w - b.w);
}


__global__
void init_vec(int nels, float4* __restrict__ d_vec1)
{
	int Idx = threadIdx.x + blockIdx.x * blockDim.x;
	int i= Idx*4;
	d_vec1[Idx].x = i;
	d_vec1[Idx].y = i+1;
	d_vec1[Idx].z = i+2;
	d_vec1[Idx].w = i+3;



}

__global__
void multi_vec2(int nels,int n_row1,int n_col1,int n_row2,int n_col2,float4* __restrict__ res_vec,
								float* __restrict__ d_vec1,float* __restrict__ d_vec2)
{
	int Idx = threadIdx.x + blockIdx.x * blockDim.x;
	int i= Idx*4;
	int r_res,c_res;
	r_res=n_row1;
	c_res=n_row2*n_col2;
	if(i<(r_res*c_res)){
		int c= ((int)(i/c_res))*n_row1 + ((int)(i%n_col1))%n_col1;
		int j= ((int)(((int)(i%c_res))/n_row2) + (((int)(i%c_res))%n_row2)*n_col2);
		res_vec[Idx].x=d_vec1[c]*d_vec2[j];
		int c1= ((int)((i+1)/c_res))*n_row1 + ((int)((i+1)%n_col1))%n_col1;
		int j1= ((int)(((int)((i+1)%c_res))/n_row2) + (((int)((i+1)%c_res))%n_row2)*n_col2);
		res_vec[Idx].y=d_vec1[c1]*d_vec2[j1];
		int c2= ((int)((i+2)/c_res))*n_row1 + ((int)((i+2)%n_col1))%n_col1;
		int j2= ((int)(((int)((i+2)%c_res))/n_row2) + (((int)((i+2)%c_res))%n_row2)*n_col2);
		res_vec[Idx].z=d_vec1[c2]*d_vec2[j2];
		int c3= ((int)((i+3)/c_res))*n_row1 + ((int)((i+3)%n_col1))%n_col1;
		int j3= ((int)(((int)((i+3)%c_res))/n_row2) + (((int)((i+3)%c_res))%n_row2)*n_col2);
		res_vec[Idx].w=d_vec1[c3]*d_vec2[j3];
	}
}

__global__
void multi_vec(int nels,int n_row1,int n_col1,int n_row2,int n_col2,float4* __restrict__ res_vec,
								float4* __restrict__ d_vec1,float4* __restrict__ d_vec2)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int r_res,c_res;
	r_res=n_row1;
	c_res=n_row2*n_col2;
	if(i<(r_res*c_res)){
		int c= ((int)(i/c_res))*n_col1 + ((int)(i%n_col1))%n_col1;
		int j= ((int)(((int)(i%c_res))/n_row2) + (((int)(i%c_res))%n_row2)*n_col2);

			res_vec[i].x=d_vec1[c].x*d_vec2[j].x;
			res_vec[i].y=d_vec1[c].y*d_vec2[j].y;
			res_vec[i].z=d_vec1[c].z*d_vec2[j].z;
			res_vec[i].w=d_vec1[c].w*d_vec2[j].w;
	}
}

__global__
void scalareMatrice( float4* __restrict__ res_vec,float scalar,float4* __restrict__ d_vec)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	res_vec[i].x=d_vec[i].x*scalar;
	res_vec[i].y=d_vec[i].y*scalar;
	res_vec[i].z=d_vec[i].z*scalar;
	res_vec[i].w=d_vec[i].w*scalar;
}


__global__
void reduction_row2(int nels,int l_elem,float4* res_vec, float4*  d_vec1)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	const float4 noels = make_float4(0.0, 0.0, 0.0, 0.0);
	const int nquarts = nels*4;
	const int elem=nels/l_elem;
	//int i=idx*l_elem;
	int i0 = idx;
	int i1 = idx + 1;
	int i2 = idx + 2;
	int i3 = idx + 3;
		 __syncthreads();
	float4 r0;
	if(l_elem >= 4){
		r0=d_vec1[i0];
	}
	else r0= noels;
	float4 r1;
	if(l_elem >= 8){
		r1=d_vec1[i1];

	}
	else r1= noels;

	float4 r2;
	if(l_elem >= 12){
		r2=d_vec1[i2];
	}
	else r2= noels;
	float4 r3;
	if(l_elem >= 16){
		r3=d_vec1[i3];
	}
	else r3= noels;

	float4 v = (r0 + r1) + (r2 + r3);

	if (idx < nels){
		if(idx%4==0)
			res_vec[idx].x = (v.x + v.y) + (v.z + v.w);
		if(idx%4==1)
			res_vec[idx].y = (v.x + v.y) + (v.z + v.w);
		if(idx%4==2)
			res_vec[idx].z = (v.x + v.y) + (v.z + v.w);
		if(idx%4==3)
			res_vec[idx].w = (v.x + v.y) + (v.z + v.w);
	}
}

__global__
void reduction_row(int nels,int l_elem,float4* res_vec, float4*  d_vec1)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	const float4 noels = make_float4(0.0, 0.0, 0.0, 0.0);
	const int nquarts = nels*4;
	const int elem=nels/l_elem;
	int i=idx*(l_elem/4);
	int i0 = i;
	int i1 = i + 1;
	int i2 = i + 2;
	int i3 = i + 3;
		 __syncthreads();
	float4 r0;
	if(l_elem >= 4){
		r0=d_vec1[i0];
	}
	else r0= noels;
	float4 r1;
	if(l_elem >= 8){
		r1=d_vec1[i1];

	}
	else r1= noels;

	float4 r2;
	if(l_elem >= 12){
		r2=d_vec1[i2];
	}
	else r2= noels;
	float4 r3;
	if(l_elem >= 16){
		r3=d_vec1[i3];
	}
	else r3= noels;

	float4 v = (r0 + r1) + (r2 + r3);


	if (idx < nels){
		int x= idx/4;
		if(idx%4==0)
			res_vec[x].x = (v.x + v.y) + (v.z + v.w);
		if(idx%4==1)
			res_vec[x].y = (v.x + v.y) + (v.z + v.w);
		if(idx%4==2)
			res_vec[x].z = (v.x + v.y) + (v.z + v.w);
		if(idx%4==3)
			res_vec[x].w = (v.x + v.y) + (v.z + v.w);
	}
}


__global__
void transpose(int nrow,int ncols, float4* __restrict__ res_vec, float4* __restrict__ d_vec1)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int c =i%ncols;
	int r=i/ncols;
	int l_in = r*ncols + c;
	int l_out = c * nrow + r;
	res_vec[l_out].x = d_vec1[l_in].x;
	res_vec[l_out].y = d_vec1[l_in].y;
	res_vec[l_out].z = d_vec1[l_in].z;
	res_vec[l_out].w = d_vec1[l_in].w;

}


__global__
void vecsum(int nels, float4* __restrict__ res_vec, float4* __restrict__ d_vec1, float4* __restrict__ d_vec2)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;;

	res_vec[i] =d_vec1[i]+d_vec2[i];

}

__global__
void vecdif(int nels, float4* __restrict__ res_vec, float4* __restrict__ d_vec1, float4* __restrict__ d_vec2)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	res_vec[i] =d_vec1[i]-d_vec2[i];
}
void stampa(float* matrice,int m){
  int i,j;
	printf("\n");
  for(i=0;i<m;i++){
      printf("%f  ",matrice[i]);
			printf("\n");
	}

}

int main(int argc, char *argv[]){
  float4* matriceA;
  float4* matriceB;
  float4* matriceX;
	float4* pk;
	float4* trasposta;
	float4* prodotto;
	float4* somma;
	float4* res;
	float4* den;
	float4* res0;
	float4* res1;
	float4* res2;
	float4* red_den;
  float* matrice;
	float4* scalar;
	float4* num;
	float4* deno;
  float ak;
  int nels;
  if (argc != 2) {
    error("syntax: vecsum nels v");
  }

  int N = atoi(argv[1]);
  if (N < 0) {
    error("N < 0");
  }
	int M=1;

  nels=N*N;
  size_t memsize = nels*sizeof(float);
  hipError_t err;
  err = hipMalloc((void**)&matriceA, memsize);
  cuda_check(err, "alloc matriceA");
  err = hipMalloc((void**)&matriceB, N*M*sizeof(float));
  cuda_check(err, "alloc matriceB");
  err = hipMalloc((void**)&matriceX, N*sizeof(float));
  cuda_check(err, "alloc matriceX");
  err = hipHostMalloc(&matrice, N*N*sizeof(float), hipHostMallocDefault);
	cuda_check(err, "alloc matrice");
	err = hipHostMalloc(&num, M*sizeof(float), hipHostMallocDefault);
	cuda_check(err, "alloc matrice");
	err = hipHostMalloc(&deno, M*sizeof(float), hipHostMallocDefault);
	cuda_check(err, "alloc matrice");
	err = hipMalloc((void**)&somma,nels*M*sizeof(float));
	cuda_check(err, "alloc somma");
	err = hipMalloc((void**)&res,M*N*N*sizeof(float));
	cuda_check(err, "alloc res");
	err = hipMalloc((void**)&res0,N*M*N*sizeof(float));
	cuda_check(err, "alloc res0");
	err = hipMalloc((void**)&prodotto,M*N*N*sizeof(float));
	cuda_check(err, "alloc prodotto");
	err = hipMalloc((void**)&res1,M*N*sizeof(float));
	cuda_check(err, "alloc res1");
	err = hipMalloc((void**)&res2,M*N*N*sizeof(float));
	cuda_check(err, "alloc res2");
	err = hipMalloc((void**)&pk,M*N*sizeof(float));
	cuda_check(err, "alloc pk");
	err = hipMalloc((void**)&trasposta,M*N*sizeof(float));
	cuda_check(err, "alloc trasposta	");
	err = hipMalloc((void**)&den,M*N*sizeof(float));
	cuda_check(err, "alloc den");
	err = hipMalloc((void**)&red_den,M*N*sizeof(float));
	cuda_check(err, "alloc den");
	err = hipMalloc((void**)&scalar,M*N*sizeof(float));
	cuda_check(err, "alloc scalar");

	hipEvent_t pre_init, post_init, pre_sum, post_sum,pre_prodotto,post_prodotto,
							pre_transpose,post_transpose,pre_scalar_matrice,post_scalar_matrice,pre_vecsum,post_vecsum,
							pre_vecdif,post_vecdif;


	err = hipEventCreateWithFlags(&pre_init, 0);
	cuda_check(err, "create pre_init");
	err = hipEventCreateWithFlags(&pre_prodotto, 0);
	cuda_check(err, "create pre_sum");
	err = hipEventCreateWithFlags(&pre_transpose, 0);
	cuda_check(err, "create pre_traspose");
	err = hipEventCreateWithFlags(&pre_scalar_matrice, 0);
	cuda_check(err, "create pre_scalar_matrice");
	err = hipEventCreateWithFlags(&pre_vecdif, 0);
	cuda_check(err, "create pre_vecdif");
	err = hipEventCreateWithFlags(&pre_vecsum, 0);
	cuda_check(err, "create pre_vecsum");

	err = hipEventCreateWithFlags(&post_init, 0);
	cuda_check(err, "create post_init");
	err = hipEventCreateWithFlags(&post_prodotto, 0);
	cuda_check(err, "create post_sum");
	err = hipEventCreateWithFlags(&post_transpose, 0);
	cuda_check(err, "create post_traspose");
	err = hipEventCreateWithFlags(&post_scalar_matrice, 0);
	cuda_check(err, "create post_scalar_matrice");
	err = hipEventCreateWithFlags(&post_vecdif, 0);
	cuda_check(err, "create post_vecdif");
	err = hipEventCreateWithFlags(&post_vecsum, 0);
	cuda_check(err, "create post_vecsum");
  const int blockSize = 1024;
	int numBlocks = (nels/4 + blockSize - 1)/blockSize;


	hipEventRecord(pre_init);
	init_vec<<<blockSize,numBlocks>>>(nels, matriceA);
	hipEventRecord(post_init);

	numBlocks = (M*N/4 + blockSize - 1)/blockSize;
  init_vec<<<blockSize, numBlocks>>>(M*N, matriceB);

	init_vec<<<blockSize, numBlocks>>>(M*N, matriceX);
	int i;

	//calcolo i parametri della riduzione
	int THREAD_LOAD=0;
	float n = N;
	while (n > 1) {
		n/=4;
		if(n==1){
			THREAD_LOAD=4;
		}
	}
	n = N;
	while (n > 1) {

		n/=8;
		if(n==1){
			THREAD_LOAD=8;
		}
	}
	n=N;
	while (n > 1) {
		n/=12;
		if(n==1){
			THREAD_LOAD=12;
		}
	}
	n=N;
	while (n > 1) {
		n/=16;
		if(n==1){
			THREAD_LOAD=16;
		}
	}
	if(THREAD_LOAD==0){
		printf("Errore N deve essere una potenza di 4,8,12,16");
		exit(0);
	}
	int j;
	int c=N;
	float* temp;
	float runtime_red_ms;
	int lr=0;
	int log=N*N;
	while(log>N){
		++lr;
		log=log/THREAD_LOAD;
	}

	hipEvent_t pre_red[lr], post_red[lr];
	//inizializzo gli eventi per la riduzione
	for(i=0;i<lr;i++){
		err = hipEventCreateWithFlags(&(pre_red[i]), 0);
		cuda_check(err, "create pre_red");

		err = hipEventCreateWithFlags(&(post_red[i]), 0);
		cuda_check(err, "create post_red");
	}
	for(i=0;i<1;i++){
		numBlocks = (nels/4 + blockSize - 1)/blockSize;
		hipEventRecord(pre_prodotto);
		multi_vec<<<blockSize, numBlocks>>>(nels*M/4,N,N/4,N/4,M,somma,matriceA,matriceX);
		hipEventRecord(post_prodotto);

		c=N*N;
		int nels_red=0;
		int cont=0;
		while(c>N){
			c/=THREAD_LOAD;
			nels_red+=c;
			numBlocks = (c + blockSize - 1)/blockSize;
			hipEventRecord(pre_red[cont]);
			reduction_row<<<blockSize, numBlocks>>>(c,THREAD_LOAD,res0,somma);
			hipEventRecord(post_red[cont]);
			err = hipMemcpy(somma, res0, c*sizeof(float4), hipMemcpyDeviceToDevice);
			cuda_check(err, "cpy");
			cont++;
		}




		printf("%d %d\n",lr,nels_red );
		numBlocks = ((N*M)/4 + blockSize - 1)/blockSize;
		hipEventRecord(pre_vecdif);
		vecdif<<<blockSize, numBlocks>>>(N*M,pk,matriceB,res0);
		hipEventRecord(post_vecdif);


		numBlocks = (N*N/4 + blockSize - 1)/blockSize;
		hipEventRecord(pre_transpose);
		transpose<<<blockSize, numBlocks>>>(N,M,trasposta,pk);
		hipEventRecord(post_transpose);

		numBlocks = ((M*N)/4 + blockSize - 1)/blockSize;
		multi_vec<<<blockSize, numBlocks>>>(N*M/4,M,N/4,N/4,M,prodotto,trasposta,pk);
		c=N;
		while (c>1) {
			c/=THREAD_LOAD;
			numBlocks = (c + blockSize - 1)/blockSize;
			reduction_row<<<blockSize, numBlocks>>>(c,THREAD_LOAD,res1,prodotto);
			err = hipMemcpy(prodotto, res1, c*sizeof(float), hipMemcpyDeviceToDevice);
			cuda_check(err, "cpy");
		}

		numBlocks = ((M*N*N*M)/4 + blockSize - 1)/blockSize;
		multi_vec2<<<blockSize, numBlocks>>>(M*N*N*M/4,M,N,N,N,res,(float*)trasposta,(float*)matriceA);
		c=N*N;
		while (c>N) {
			c/=THREAD_LOAD;
			numBlocks = (c + blockSize - 1)/blockSize;
			reduction_row<<<blockSize, numBlocks>>>(c,THREAD_LOAD,res2,res);
			err = hipMemcpy(res, res2, c*sizeof(float), hipMemcpyDeviceToDevice);
			cuda_check(err, "cpy");
		}



		numBlocks = ((N*N)/4 + blockSize - 1)/blockSize;
		multi_vec<<<blockSize, numBlocks>>>(N*N/4	,M,N/4,N/4,M,den,res2,pk);
		c=N;
		while (c>1) {
			c/=THREAD_LOAD;
			numBlocks = (c + blockSize - 1)/blockSize;
			reduction_row<<<blockSize, numBlocks>>>(c,THREAD_LOAD,red_den,den);
			err = hipMemcpy(den, red_den, c*sizeof(float), hipMemcpyDeviceToDevice);
			cuda_check(err, "cpy");
		}
		err = hipMemcpy(num, res1, 1*sizeof(float), hipMemcpyDeviceToHost);
		err = hipMemcpy(deno, red_den, 1*sizeof(float), hipMemcpyDeviceToHost);
		ak=num[0].x/deno[0].x;
		printf("%f\n",ak );
		numBlocks = (N/4 + blockSize - 1)/blockSize;
		hipEventRecord(pre_scalar_matrice);
		scalareMatrice<<<blockSize, numBlocks>>>(scalar,ak,pk);
		hipEventRecord(post_scalar_matrice);

		numBlocks = ((N*M)/4 + blockSize - 1)/blockSize;
		hipEventRecord(pre_vecsum);
		vecsum<<<blockSize, numBlocks>>>(N*M,matriceX,matriceX,scalar);
		hipEventRecord(post_vecsum);

		err = hipMemcpy(matrice, matriceX, M*N*sizeof(float), hipMemcpyDeviceToHost);
		cuda_check(err, "create mem");
	  stampa(matrice,M*N);



		float runtime_init_ms, runtime_prodotto_ms, runtime_red_ms,runtime_transpose_ms,runtime_scalar_matrice_ms,
					runtime_vecdif_ms,runtime_vecsum_ms,runtime_red_count_ms;
		err = hipEventElapsedTime(&runtime_init_ms, pre_init, post_init);
		cuda_check(err, "elapsed time init");
		err = hipEventElapsedTime(&runtime_prodotto_ms, pre_prodotto, post_prodotto);
		cuda_check(err, "elapsed time prodotto");
		runtime_red_count_ms=0;
		for(j=0;j<lr;j++){

			err = hipEventElapsedTime(&runtime_red_ms, pre_red[j], post_red[j]);
			cuda_check(err, "elapsed time reduction");
			runtime_red_count_ms+=runtime_red_ms;

		}
		err = hipEventElapsedTime(&runtime_transpose_ms, pre_transpose, post_transpose);
		cuda_check(err, "elapsed time traspose");
		err = hipEventElapsedTime(&runtime_scalar_matrice_ms, pre_scalar_matrice, post_scalar_matrice);
		cuda_check(err, "elapsed time scalar_matrice");
		err = hipEventElapsedTime(&runtime_vecdif_ms, pre_vecdif, post_vecdif);
		cuda_check(err, "elapsed time vecdif");
		err = hipEventElapsedTime(&runtime_vecsum_ms, pre_vecsum, post_vecsum);
		cuda_check(err, "elapsed time vecsum");


		printf("init: runtime %.4gms, %.4g GE/s, %.4g GB/s\n",
			runtime_init_ms, nels/runtime_init_ms/1.0e6, memsize/runtime_init_ms/1.0e6);
		printf("prodotto: runtime %.4gms, %.4g GE/s, %.4g GB/s\n",
			runtime_prodotto_ms, nels/runtime_prodotto_ms/1.0e6, memsize/runtime_prodotto_ms/1.0e6);
		printf("reduction: runtime %.4gms, %.4g GE/s, %.4g GB/s\n",
			runtime_red_count_ms, nels_red/runtime_red_count_ms/1.0e6, (nels_red*sizeof(float))/runtime_red_count_ms/1.0e6);
		printf("transpose: runtime %.4gms, %.4g GE/s, %.4g GB/s\n",
			runtime_transpose_ms, N/runtime_transpose_ms/1.0e6, (N*sizeof(float))/runtime_transpose_ms/1.0e6);
		printf("scalareMatrice: runtime %.4gms, %.4g GE/s, %.4g GB/s\n",
			runtime_scalar_matrice_ms, N/runtime_scalar_matrice_ms/1.0e6, (N*sizeof(float))/runtime_scalar_matrice_ms/1.0e6);
		printf("vecdif: runtime %.4gms, %.4g GE/s, %.4g GB/s\n",
			runtime_vecdif_ms, N/runtime_vecdif_ms/1.0e6, (N*sizeof(float))/runtime_vecdif_ms/1.0e6);
		printf("vecsum: runtime %.4gms, %.4g GE/s, %.4g GB/s\n",
				runtime_vecsum_ms, N/runtime_vecsum_ms/1.0e6, (N*sizeof(float))/runtime_vecsum_ms/1.0e6);

	}
  hipFree(matriceA);
	hipHostFree(matrice);
	hipFree(somma);
	hipFree(res);
	hipFree(pk);
	hipFree(trasposta);
	hipFree(prodotto);
	hipFree(den);
	hipFree(res0);
	hipFree(res1);
	hipFree(res2);
	hipFree(red_den);
	hipFree(scalar);
	hipFree(matriceB);
	hipFree(matriceX);
	hipHostFree(num);
	hipHostFree(deno);

}
